#include "hip/hip_runtime.h"
﻿#include "mesh.cuh"
#include "triangle.cuh"
#include "vec3.cuh"
#include "object.cuh"
#include "hitresult.cuh"
#include "ray.cuh"

#include "hip/hip_runtime.h"
#include "stdio.h"
#include "dxhook/mainHook.h"
constexpr float kEpsilon = 1e-8;
constexpr float MAX_FLOAT = 1000000.0f;

#define MOLLER_TRUMBORE
// #define CULLING



__host__ __device__ static bool rayTriangleIntersect(
    const Tracer::vec3& orig, const Tracer::vec3& dir,
    const Tracer::vec3& v0, const Tracer::vec3& v1, const Tracer::vec3& v2,
    float& t, float& u, float& v)
{
    using Tracer::vec3;
    /*
#ifdef MOLLER_TRUMBORE 
    vec3 v0v1 = v1 - v0;
    vec3 v0v2 = v2 - v0;
    vec3 pvec = cross(dir, v0v2);
    float det = (float)dot(v0v1, pvec);
#ifdef CULLING 
    // if the determinant is negative the triangle is backfacing
    // if the determinant is close to 0, the ray misses the triangle
    if (!(det < kEpsilon)) return false;
#else 
    // ray and triangle are parallel if det is close to 0
    if (fabs(det) < kEpsilon) return false;
#endif 
    float invDet = 1.f / det;

    vec3 tvec = orig - v0;
    u = dot(tvec, pvec) * invDet;
    if (u < 0 || u > 1) return false;

    vec3 qvec = cross(tvec, v0v1);
    v = dot(dir, qvec) * invDet;
    if (v < 0 || u + v > 1) return false;

    t = dot(v0v2, qvec) * invDet;

    return true;
#else 
    // compute plane's normal
    Tracer::vec3 v0v1 = v1 - v0;
    Tracer::vec3 v0v2 = v2 - v0;
    // no need to normalize
    Tracer::vec3 N = v0v1.cross(v0v2); // N 
    float denom = N.dot(N);

    // Step 1: finding P

    // check if ray and plane are parallel ?
    float NdotRayDirection = N.dot(dir);
    if (fabs(NdotRayDirection) < kEpsilon) // almost 0 
        return false; // they are parallel so they don't intersect ! 

    // compute d parameter using equation 2
    float d = N.dot(v0);

    // compute t (equation 3)
    t = (N.dot(orig) + d) / NdotRayDirection;
    // check if the triangle is in behind the ray
    if (t < 0) return false; // the triangle is behind 

    // compute the intersection point using equation 1
    Tracer::vec3 P = orig + (dir * t);

    // Step 2: inside-outside test
    Tracer::vec3 C(0, 0, 0); // vector perpendicular to triangle's plane 

    // edge 0
    Tracer::vec3 edge0 = v1 - v0;
    Tracer::vec3  vp0 = P - v0;
    C = edge0.cross(vp0);
    if (N.dot(C) < 0) return false; // P is on the right side 

    // edge 1
    Tracer::vec3  edge1 = v2 - v1;
    Tracer::vec3  vp1 = P - v1;
    C = edge1.cross(vp1);
    if ((u = N.dot(C)) < 0)  return false; // P is on the right side 

    // edge 2
    Tracer::vec3  edge2 = v0 - v2;
    Tracer::vec3  vp2 = P - v2;
    C = edge2.cross(vp2);
    if ((v = N.dot(C)) < 0) return false; // P is on the right side; 

    u /= denom;
    v /= denom;

    return true; // this ray hits the triangle 
#endif 
*/
    const vec3 edge1 = v1 - v0;
    const vec3 edge2 = v2 - v0;

    const vec3 h = cross(dir, edge2);
    const float a = dot(edge1, h);
    if (a > -kEpsilon && a < kEpsilon) return false;

    const float f = 1.f / a;
    const vec3 s = orig - v0;
    u = f * dot(s, h);
    if (u < 0 || u > 1) return false;

    const vec3 q = cross(s, edge1);
    v = f * dot(dir, q);
    if (v < 0 || u + v > 1) return false;

    t = f * dot(edge2, q);
    if (t > kEpsilon) return true;

    return false;
}

#define MAX_TRIANGLES 9000
#define checkCudaErrors(val) DXHook::check_cuda( (val), #val, __FILE__, __LINE__ )
#define min(a,b) ((a)<(b)?(a):(b))
#define max(a,b) ((a)>(b)?(a):(b))

__device__ bool done = false;

static inline void swap(float a, float b) {
    float temp = a;
    a = b;
    b = temp;
}

namespace Tracer {
	__host__ __device__ Mesh::Mesh() {
		size = 0;
        triBuffer = new Triangle*[MAX_TRIANGLES];
        minV = vec3(0, 0, 0);
        maxV = vec3(0, 0, 0);

	}

    __host__ __device__ Mesh::~Mesh() {
        for (int i = 0; i < size; i++) {
            delete triBuffer[i];
        }

        delete triBuffer;
    }
	__host__ __device__ void Mesh::InsertTri(vec3 v1, vec3 v2, vec3 v3, float u1, float u2, float u3, float vt1, float vt2, float vt3) {
        Triangle* theTri = new Triangle(v1, v2, v3, u1, u2, u3, vt1, vt2, vt3);

        if ((size + 1) >= MAX_TRIANGLES) {
            printf("MAX TRIANGLES LIMIT REACHED!!!!");
            return;
        }
        else {
            printf("[gpu]: Triangle inserted on GPU!.. i think v1: %.2f, %.2f, %2.f\n", v1.x(), v1.y(), v1.z());
        }

        triBuffer[size++] = theTri;
	}

    __host__ __device__ void Mesh::ComputeAccel(vec3 newMin, vec3 newMax) {
        // bounds[0] == min
        // bounds[1] == max
        
        minV = newMin;
        maxV = newMax;
        
       
        printf("min: %.2f, %.2f, %.2f\nmax: %.2f, %.2f, %.2f\n", minV.x(), minV.y(), minV.z(), maxV.x(), maxV.y(), maxV.z());
        
    }

    __host__ __device__ bool Mesh::anyHit(const Ray& ray) {
        
        vec3 nLocal = ray.invorig - ray.invdir * (minV + maxV) / 2.f;

        vec3 k = vec3(abs(ray.invdir.x()), abs(ray.invdir.y()), abs(ray.invdir.z())) * (maxV - minV) / 2.f;
        vec3 t1 = -nLocal - k;
        vec3 t2 = -nLocal + k;

        double tNear = max(max(t1.x(), t1.y()), t1.z());
        double tFar = min(min(t2.x(), t2.y()), t2.z());


        return !(tNear > tFar || tFar < 0);
        

        /*
        vec3 tMin = (minV - ray.origin) / ray.direction;
        vec3 tMax = (maxV - ray.origin) / ray.direction;
        vec3 t1 = min(tMin, tMax);
        vec3 t2 = max(tMin, tMax);
        float tNear = max(max(t1.x(), t1.y()), t1.z());
        float tFar = min(min(t2.x(), t2.y()), t2.z());

        tMaxA = tFar;

        return tNear > tFar;
        */

        //return true;
    }

    __host__ __device__ bool Mesh::tryHit(const Ray& ray, HitResult& closestHit) {
        bool bHit = false;

        for (int i = 0; i < size; i++) {
            Triangle* triHere = triBuffer[i];
   
            float t = 0.f;
            float u = 0.f;
            float v = 0.f;

            if (rayTriangleIntersect(ray.origin, ray.direction, triHere->v1, triHere->v2, triHere->v3, t, u, v) && t > kEpsilon && t < closestHit.t) {
                closestHit.t = t;
                closestHit.u = (1.f - u - v) * triHere->u1 + u * triHere->u2 + v * triHere->u3;
                closestHit.v = (1.f - u - v) * triHere->vt1 + u * triHere->vt2 + v * triHere->vt3;
                
                // Account for >1 and <1 UVs
                closestHit.u -= floorf(u);
                closestHit.v -= floorf(v);

                closestHit.HitNormal = triHere->normal;
                closestHit.objId = objectID;

                bHit = true;
            }
        }

        return bHit;
	}
}