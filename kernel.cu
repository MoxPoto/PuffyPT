#include "hip/hip_runtime.h"
#include <GarrysMod/Lua/Interface.h>
#include <Windows.h>

#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"
#include "hip/hip_math_constants.h"

#include <stdio.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <chrono>
#define _USE_MATH_DEFINES
#include <cmath>

#include <util/macros.h>
#include <brdfs/lambert.cuh>
#include <brdfs/specular.cuh>
#include <brdfs/refraction.cuh>
#include <images/hdriUtility.cuh>

#include <dxhook/mainHook.h>
#include <postprocess/mainDenoiser.cuh>
#include <cpugpu/objects.cuh>
#include <synchronization/syncMain.cuh>

#include <pathtracer.cuh>

#define STB_IMAGE_IMPLEMENTATION
#include <vendor/stb_image.h>

#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <glm/mat3x3.hpp>
#include <glm/vec3.hpp>
#include <glm/gtc/matrix_transform.hpp>

#define WIDTH 480
#define HEIGHT 270
#define checkCudaErrors(val) DXHook::check_cuda( (val), #val, __FILE__, __LINE__ )
#define HDRI_LOCATION "C:\\pathtracer\\hdrs\\shanghai_bund_1k.hdr"
#define HDRI_FOLDER "C:\\pathtracer\\hdrs"
#define HDRI_RESX 2048
#define HDRI_RESY 1024

void DXHook::check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cout << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n" << "CUDA_ERROR_STRING: " << hipGetErrorString(result) << "\n" <<
            hipGetErrorName(result) << "\n";
    }
}

__global__ void DXHook::render(DXHook::RenderOptions options) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= options.max_x) || (j >= options.max_y)) return;
    int pixel_index = j * options.max_x * 3 + i * 3;
    int random_idx = j * options.max_x + i;

    hiprandState local_rand_state = options.rand_state[random_idx];

    hiprand_init(options.frameCount * options.max_x * options.max_y + j * options.max_x + i, 1, 0, &local_rand_state);

    Post::GBuffer* gbuffer = ((options.gbufferPtr + random_idx)); // serves as a gbuffer access index too!!

    vec3 frameColor;

    float DISTANCE = 1.f;

    float coeff = DISTANCE * tan((options.fov / 2) * (M_PI / 180)) * 2;
    vec3 camOrigin = vec3(
        DISTANCE,
        (static_cast<float>(options.max_x - i) / static_cast<float>(options.max_x - 1) - 0.5) * coeff,
        (coeff / static_cast<float>(options.max_x)) * static_cast<float>(options.max_y - j) - 0.5 * (coeff / static_cast<double>(options.max_x)) * static_cast<double>(options.max_y - 1)
    );
    vec3 dir = unit_vector(camOrigin);
    // NOT MY CODE!! https://github.com/100PXSquared/public-starfalls/tree/master/raytracer

    glm::mat4 rotationMat(1.f);

    // X is roll..
    // Z is yaw
    // so Y is pitch!! YAY!! SOMETHING SORT OF SENSIBLE!!

    rotationMat = glm::rotate(rotationMat, glm::radians(-options.cameraDir.x()), glm::vec3(0, 1, 0));
    rotationMat = glm::rotate(rotationMat, glm::radians(options.cameraDir.y()), glm::vec3(0, 0, 1));

    glm::vec4 preVec = rotationMat * glm::vec4(dir.x(), dir.y(), dir.z(), 0);

    dir = vec3(preVec.x, preVec.y, preVec.z);

    vec3 origin(options.x, options.y, options.z);

    Ray ourRay(origin, dir);

    HitResult result;
    Object* hitObject = traceScene(options.count, options.world, ourRay, result);

    int samples = options.samples;
    int max_depth = options.max_depth;

    // while we're here, let's update our HDRI's brightness as told to by the Host
    options.hdri->brightness = options.hdriBrightness;
    
    if (hitObject != NULL) {
        Ray newRay = ourRay;
        newRay.origin = newRay.origin + (result.HitNormal * 0.001f);

        vec3 indirect = pathtrace(&options, newRay, &local_rand_state);

        frameColor = indirect;
    }
    else {
        if (options.doSky) {
            vec3 skyColor = genSkyColor(options.hdri, options.skyInfo, options.hdriData, dir);

            frameColor = skyColor;
        }
    }
    
    if (hitObject != NULL) {
        gbuffer->albedo = hitObject->GetColor(result);
        gbuffer->normal = result.HitNormal;
        gbuffer->objectID = result.objId;
        gbuffer->brdfType = hitObject->matType;
    }
    
    gbuffer->position = result.HitPos;
    gbuffer->depth = result.t;
    gbuffer->diffuse = frameColor;
    gbuffer->isSky = (hitObject == NULL);
    
    vec3 previousFrame = vec3(options.frameBuffer[pixel_index + 0], options.frameBuffer[pixel_index + 1], options.frameBuffer[pixel_index + 2]);
    vec3 accumulated = (frameColor + previousFrame * options.frameCount) / (options.frameCount + 1);

    // Accumulation can give way to NaN frames which result in black dots
    // so, check if our new pixel is nan--if it is, then restore old frame

    if (isnan(accumulated.x()) || isnan(accumulated.y()) || isnan(accumulated.z()))
        accumulated = previousFrame;

    options.frameBuffer[pixel_index + 0] = accumulated.r();
    options.frameBuffer[pixel_index + 1] = accumulated.g();
    options.frameBuffer[pixel_index + 2] = accumulated.b();
}

__global__ void DXHook::registerRands(int max_x, int max_y, hiprandState* rand_state, Post::GBuffer* gbufferData) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984 + pixel_index, pixel_index, 0, &rand_state[pixel_index]);
    // lets also initialize our GBuffers
    Post::GBuffer myBuffer;

    *(gbufferData + pixel_index) = myBuffer;
}

__global__ void freeMem(Object** world, vec3* origin, int worldCount) {
    for (int i = 0; i < worldCount; i++) {
        delete* (world + i);
    }

    delete origin;
}

GMOD_MODULE_OPEN()
{
    hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024000 * 10);

    AllocConsole();
    FILE* pFile = nullptr;

    freopen_s(&pFile, "CONOUT$", "w", stdout); // cursed way to redirect stdout to our own console

    DXHook::lastTime = std::chrono::high_resolution_clock::now();

    HOST_DEBUG("Querying device..");
    int ourDeviceID;
    checkCudaErrors(hipGetDevice(&ourDeviceID));

    HOST_DEBUG("Got device!");
    hipDeviceProp_t properties;

    checkCudaErrors(hipGetDeviceProperties(&properties, ourDeviceID));

    HOST_DEBUG("Got properties..");

    HOST_DEBUG("Using GPU %s\n", properties.name);
    HOST_DEBUG("Is integrated: %d\n", properties.integrated);
    HOST_DEBUG("Max threads per block: %d\n", properties.maxThreadsPerBlock);
    HOST_DEBUG("GPU's MP count: %d\n", properties.multiProcessorCount);
    HOST_DEBUG("Major: %d, Minor: %d", properties.major, properties.minor);

    HOST_DEBUG("Starting memory allocation for GPU");

    int num_pixels = WIDTH * HEIGHT;
    size_t fb_size = 3 * num_pixels * sizeof(float);
    size_t world_size = 260 * sizeof(Object*);
    size_t origin_size = sizeof(vec3*);
    size_t gbuffer_size = num_pixels * sizeof(Post::GBuffer);
    size_t imageSize = 3 * (HDRI_RESX * HDRI_RESY) * sizeof(float);
    size_t hdriSize = sizeof(HDRI*);

    HOST_DEBUG("Calculated sizes..");

    checkCudaErrors(hipMallocManaged((void**)&DXHook::fb, fb_size));
    checkCudaErrors(hipMallocManaged((void**)&DXHook::postFB, fb_size));
    checkCudaErrors(hipMallocManaged((void**)&DXHook::bloomFB, fb_size));
    checkCudaErrors(hipMallocManaged((void**)&DXHook::blurFB, fb_size));

    checkCudaErrors(hipMallocManaged((void**)&DXHook::world, world_size));
    checkCudaErrors(hipMallocManaged((void**)&DXHook::origin, origin_size));

    checkCudaErrors(hipMalloc((void**)&DXHook::gbufferData, gbuffer_size));
    checkCudaErrors(hipMalloc((void**)&DXHook::d_rand_state, num_pixels * sizeof(hiprandState)));
    checkCudaErrors(hipMallocManaged((void**)&DXHook::hdriData, imageSize));
    checkCudaErrors(hipMallocManaged((void**)&DXHook::mainHDRI, hdriSize));

    HOST_DEBUG("Allocated all memory");

    HOST_DEBUG("Reading HDRI from disk..");
    
    bool correctLoad = LoadHDRI(HDRI_LOCATION);

    if (!correctLoad) {
        HOST_DEBUG("Loading HDRI failed! Not continuing tracer loading..");
        return 0;
    }

    FindHDRIs(HDRI_FOLDER, DXHook::hdriList, DXHook::hdriListSize);

    for (int i = 0; i < DXHook::hdriListSize; i++) {
        std::string path = DXHook::hdriList.at(i);

        if (path == HDRI_LOCATION) {
            DXHook::curHDRI = i;
            break;
        }
    }

    HOST_DEBUG("Starting random threads..");

    int warpX = 16;
    int warpY = 16; // technically can be ruled out as tiled rendering

    dim3 blocks(WIDTH / warpX + 1, HEIGHT / warpY + 1);
    dim3 threads(warpX, warpY);

    ClearFramebuffer << <blocks, threads >> > (DXHook::fb, WIDTH, HEIGHT);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    ClearFramebuffer << <blocks, threads >> > (DXHook::postFB, WIDTH, HEIGHT);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    ClearFramebuffer << <blocks, threads >> > (DXHook::bloomFB , WIDTH, HEIGHT);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    DXHook::registerRands << < blocks, threads >> > (WIDTH, HEIGHT, DXHook::d_rand_state, DXHook::gbufferData);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    HOST_DEBUG("Finished!");
    // Run all of our starting CUDA code

    HOST_DEBUG("Starting DXHook..");
    DXHook::Initialize(LUA);
    HOST_DEBUG("Finished!");

    HOST_DEBUG("Starting Synchronization Service..");
    Sync::Initialize(LUA);
    HOST_DEBUG("Finished!");

    return 0;
}

GMOD_MODULE_CLOSE() 
{
    HOST_DEBUG("Closing module!");
    HOST_DEBUG("Closing DXHook..");
    DXHook::Cleanup(LUA);
    HOST_DEBUG("Finished!");

    Sync::Deinitialize(LUA);
    HOST_DEBUG("Closed Sync..");

    HOST_DEBUG("Freeing GPU memory, closing CUDA context..");
   
    Sleep(2000);

    freeMem << <1, 1 >> > (DXHook::world, DXHook::origin, DXHook::world_count);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(DXHook::fb));
    checkCudaErrors(hipFree(DXHook::world));
    checkCudaErrors(hipFree(DXHook::d_rand_state));
    checkCudaErrors(hipFree(DXHook::origin));
    checkCudaErrors(hipFree(DXHook::gbufferData));
    checkCudaErrors(hipFree(DXHook::blurFB));
    checkCudaErrors(hipFree(DXHook::bloomFB));
    checkCudaErrors(hipFree(DXHook::postFB));

    for (std::pair<std::string, Pixel*> devPtr : deviceTextures) {
        HOST_DEBUG("Cleaning %s", devPtr.first.c_str());

        checkCudaErrors(hipFree(devPtr.second));
    }

    deviceTextures.clear();

    HOST_DEBUG("Freeing DirectX Resources..");
    DXHook::msgFont->Release();
    DXHook::pathtraceObject->Release();
    DXHook::pathtraceOutput->Release();
    HOST_DEBUG("Done!");

    hipDeviceReset();

    HOST_DEBUG("Cuda context freed, module down!");
    HOST_DEBUG("You can close this window now!");
    FreeConsole();
        
    return 0;
}
