#include "hip/hip_runtime.h"
﻿#include <GarrysMod/Lua/Interface.h>

#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include <stdio.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <chrono>
#define _USE_MATH_DEFINES
#include <cmath>

#include "ray.cuh"
#include "mesh.cuh"
#include "sphere.cuh"
#include "vec3.cuh"
#include "object.cuh"
#include "triangle.cuh"

#include "brdfs/lambert.cuh"
#include "brdfs/specular.cuh"

#include "dxhook/mainHook.h"

#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <glm/mat3x3.hpp>
#include <glm/vec3.hpp>
#include <glm/gtc/matrix_transform.hpp>

#define WIDTH 480
#define HEIGHT 270
#define checkCudaErrors(val) DXHook::check_cuda( (val), #val, __FILE__, __LINE__ )

void DXHook::check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cout << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";

    }
}

__device__ float deg2rad(const float& degree) {
    return degree * M_PI / 180.f;
}

__device__ Tracer::vec3 genSkyColor(const Tracer::vec3& dir) {
    using namespace Tracer;

    float t = 0.5f * (dir.z() + 1.0f);
    vec3 skyColor = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);

    return skyColor;
}

__device__ Tracer::Object* traceScene(int count, Tracer::Object** world, const Tracer::Ray& ray, Tracer::HitResult& output) {
    using namespace Tracer;

    float t_max = FLT_MAX;
    float minDistance = 0.001f;

    Object* hitObject = NULL;

    for (int i = 0; i < count; i++) {
        Tracer::Object* target;

        if (i == 0) {
            target = *(world);
        }
        else {
            target = *(world + i);
        }

        if (target->tryHit(ray, t_max, output) && output.t > minDistance && output.t < t_max) {
            t_max = output.t;
            hitObject = target;
        }
    }

    return hitObject;
}


__device__ Tracer::vec3 depthColor(int count, const Tracer::Ray& ray, Tracer::Object** world, hiprandState* local_rand_state, int max_depth) {
    using namespace Tracer;

    Ray cur_ray = ray;
    vec3 currentLight(1, 1, 1);
    float pdf = 1.f / (2.f * M_PI);


    for (int i = 0; i < max_depth; i++) {
        HitResult rec;
        Tracer::Object* target = traceScene(count, world, cur_ray, rec);

        if (target != NULL) {
            // set our current ray to the new formulated one (this being perfect diffuse)
            // and attenuate our color by the albedo we hit, but we also should multiply our albedo by the objects emission
            Ray new_ray(vec3(0, 0, 0), vec3(0, 0, 0));
            vec3 attenuation(0, 0, 0);

            switch (target->matType) {
                case (BRDF::Lambertian):
                    LambertBRDF::SampleWorld(rec, local_rand_state, attenuation, new_ray, target);
                    break;
                case (BRDF::Specular):
                    SpecularBRDF::SampleWorld(rec, cur_ray, attenuation, new_ray, target);
                    break;
                default:
                    break;
            }

            currentLight *= attenuation;

            cur_ray = new_ray;
            
        }
        else {
            // didnt hit, finish our depth trace by attenuating our final hit color by the sky color
            vec3 skyColor = genSkyColor(cur_ray.direction);
            
            return (currentLight * (skyColor * 0.04f)) / pdf;
        }
    }
    return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__device__ Tracer::vec3 pathtrace(int count, Tracer::Object** world, const Tracer::Ray& ray, hiprandState* local_rand_state, int samples, int max_depth) {
    using namespace Tracer;
    vec3 indirectLighting(0, 0, 0);
    vec3 directLighting(0, 0, 0); // to be done soon

    HitResult result;
    Tracer::Object* hitObject = traceScene(count, world, ray, result);

    for (int i = 0; i < samples; i++) {
        indirectLighting += depthColor(count, ray, world, local_rand_state, max_depth);
    }

    indirectLighting /= (float)samples;


    return indirectLighting;
}

__global__ void DXHook::render(float* frameBuffer, Tracer::Object** world, float x, float y, float z, float pitch, float yaw, float roll, hiprandState* rand_state, int count, float fov, int max_x, int max_y) {
    using namespace Tracer;

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x * 3 + i * 3;
    int random_idx = j * max_x + i;

    hiprandState local_rand_state = rand_state[random_idx];

    float r = 0.f;
    float g = 0.f;
    float b = 0.f;

    float DISTANCE = 1.f;

    float coeff = DISTANCE * tan((fov / 2) * (M_PI / 180)) * 2;
    vec3 camOrigin = vec3(
        DISTANCE,
        (static_cast<float>(max_x - i) / static_cast<float>(max_x - 1) - 0.5) * coeff,
        (coeff / static_cast<float>(max_x)) * static_cast<float>(max_y - j) - 0.5 * (coeff / static_cast<double>(max_x)) * static_cast<double>(max_y - 1)
    );
    vec3 dir = unit_vector(camOrigin);
    // NOT MY CODE!! https://github.com/100PXSquared/public-starfalls/tree/master/raytracer
  
    glm::mat4 rotationMat(1.f);
    // X is roll..
    // Z is yaw
    // so Y is pitch!! YAY!! SOMETHING SORT OF SENSIBLE!!

    rotationMat = glm::rotate(rotationMat, glm::radians(pitch), glm::vec3(0, 1, 0));
    rotationMat = glm::rotate(rotationMat, glm::radians(yaw), glm::vec3(0, 0, 1));
    rotationMat = glm::rotate(rotationMat, glm::radians(roll), glm::vec3(1, 0, 0));

    glm::vec4 preVec = rotationMat * glm::vec4(dir.x(), dir.y(), dir.z(), 0);
    
    dir = vec3(preVec.x, preVec.y, preVec.z);

    vec3 origin(x, y, z);

    Ray ourRay(origin, dir);

    HitResult result;
    Tracer::Object* hitObject = traceScene(count, world, ourRay, result);

    int samples = 12;
    int max_depth = 3;

    if (hitObject != NULL) {
        Ray newRay = ourRay;
        newRay.origin = newRay.origin + (result.HitNormal * 0.001f);

        vec3 indirect = pathtrace(count, world, newRay, &local_rand_state, samples, max_depth);
        indirect.clamp();

        r = sqrt(indirect.r());
        g = sqrt(indirect.g());
        b = sqrt(indirect.b());
    }
    else {
        vec3 skyColor = genSkyColor(dir);
        
        r = skyColor.r();
        g = skyColor.g();
        b = skyColor.b();
    }

    frameBuffer[pixel_index + 0] = (frameBuffer[pixel_index + 0] + r) / 2.f;
    frameBuffer[pixel_index + 1] = (frameBuffer[pixel_index + 1] + g) / 2.f;
    frameBuffer[pixel_index + 2] = (frameBuffer[pixel_index + 2] + b) / 2.f;
}

__global__ void DXHook::initMem(Tracer::Object** world, Tracer::vec3* origin) {
    using namespace Tracer; 

    origin = (new Tracer::vec3(0, 0, 0));

    *(world) = (new Tracer::Sphere(vec3(10, 0, 0), .2f));
    Tracer::Object* objOne = *(world);
    objOne->color = vec3(1, 1, 1);
    objOne->emission = 1.f;
    objOne->matType = BRDF::Specular;

    *(world + 1) = (new Tracer::Sphere(vec3(10, 0, -3.2), 3.f));
    Tracer::Object* objTwo = *(world + 1);
    objTwo->color = vec3(1.f, 0.5f, 0.5f);
    objTwo->emission = 1.f;

    *(world + 2) = (new Tracer::Sphere(vec3(11, 3, 1), 0.7f));
    Tracer::Object* objThree = *(world + 2);
    objThree->color = vec3(1.f, 1.f, 1.f);
    objThree->emission = 50.f;

}

__global__ void DXHook::registerRands(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984 + pixel_index, pixel_index, 0, &rand_state[pixel_index]);
    
}

__global__ void freeMem(Tracer::Object** world, Tracer::vec3* origin) {
    delete* (world); // to-do actually encapsulate entities in a world
    delete origin;
}

GMOD_MODULE_OPEN()
{
    int num_pixels = WIDTH * HEIGHT;
    size_t fb_size = 3 * num_pixels * sizeof(float);
    size_t world_size = 3 * sizeof(Tracer::Object*);
    size_t origin_size = sizeof(Tracer::vec3*);

    checkCudaErrors(hipMallocManaged((void**)&DXHook::fb, fb_size));
    checkCudaErrors(hipMallocManaged((void**)&DXHook::world, world_size));
    checkCudaErrors(hipMallocManaged((void**)&DXHook::origin, origin_size));

    checkCudaErrors(hipMalloc((void**)&DXHook::d_rand_state, num_pixels * sizeof(hiprandState)));

    DXHook::initMem << <1, 1 >> > (DXHook::world, DXHook::origin);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    int warpX = 16;
    int warpY = 16; // technically can be ruled out as tiled rendering

    dim3 blocks(WIDTH / warpX + 1, HEIGHT / warpY + 1);
    dim3 threads(warpX, warpY);

    DXHook::registerRands << < blocks, threads >> > (WIDTH, HEIGHT, DXHook::d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Run all of our starting CUDA code

    DXHook::Initialize(LUA);

    return 0;
}

GMOD_MODULE_CLOSE() 
{
    DXHook::Cleanup(LUA);

    Sleep(2000);

    freeMem << <1, 1 >> > (DXHook::world, DXHook::origin);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(DXHook::fb));
    checkCudaErrors(hipFree(DXHook::world));
    checkCudaErrors(hipFree(DXHook::d_rand_state));
    checkCudaErrors(hipFree(DXHook::origin));

    DXHook::quadVertexBuffer->Release();
    DXHook::msgFont->Release();
    DXHook::pathtraceObject->Release();
    DXHook::pathtraceOutput->Release();

    hipDeviceReset();

    return 0;
}
