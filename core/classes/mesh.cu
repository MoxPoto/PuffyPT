#include "hip/hip_runtime.h"
﻿#include <classes/mesh.cuh>
#include <classes/triangle.cuh>
#include <classes/vec3.cuh>
#include <classes/object.cuh>
#include <classes/hitresult.cuh>
#include <classes/ray.cuh>

#include "hip/hip_runtime.h"
#include "stdio.h"
#include <dxhook/mainHook.h>

#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <glm/mat3x3.hpp>
#include <glm/vec3.hpp>
#include <glm/gtc/matrix_transform.hpp>


constexpr float kEpsilon = 1e-8;
constexpr float MAX_FLOAT = 1000000.0f;

__host__ __device__ static bool rayTriangleIntersect(
    const vec3& orig, const vec3& dir,
    const vec3& v0, const vec3& v1, const vec3& v2,
    float& t, float& u, float& v)
{
    const vec3 edge1 = v1 - v0;
    const vec3 edge2 = v2 - v0;

    const vec3 h = cross(dir, edge2);
    const float a = dot(edge1, h);
    if (a > -kEpsilon && a < kEpsilon) return false;

    const float f = 1.f / a;
    const vec3 s = orig - v0;
    u = f * dot(s, h);
    if (u < 0 || u > 1) return false;

    const vec3 q = cross(s, edge1);
    v = f * dot(dir, q);
    if (v < 0 || u + v > 1) return false;

    t = f * dot(edge2, q);
    if (t > kEpsilon) return true;

    return false;
}

#define MAX_TRIANGLES 9000
#define checkCudaErrors(val) DXHook::check_cuda( (val), #val, __FILE__, __LINE__ )
#define min(a,b) ((a)<(b)?(a):(b))
#define max(a,b) ((a)>(b)?(a):(b))

__device__ bool done = false;

static inline void swap(float a, float b) {
    float temp = a;
    a = b;
    b = temp;
}


__host__ __device__ Mesh::Mesh() {
	size = 0;
    triBuffer = new Triangle*[MAX_TRIANGLES];
    minV = vec3(0, 0, 0);
    maxV = vec3(0, 0, 0);

}

__host__ __device__ Mesh::~Mesh() {
    for (int i = 0; i < size; i++) {
        delete triBuffer[i];
    }

    delete triBuffer;
}
__device__ void Mesh::InsertTri(const TrianglePayload& payload) {
    Triangle* theTri = new Triangle(payload);

    if ((size + 1) >= MAX_TRIANGLES) {
        printf("MAX TRIANGLES LIMIT REACHED!!!!");
        return;
    }

    triBuffer[size++] = theTri;
}

__host__ __device__ void Mesh::ComputeAccel(vec3 newMin, vec3 newMax) {
    // bounds[0] == min
    // bounds[1] == max
        
    minV = newMin;
    maxV = newMax;
        
       
    printf("min: %.2f, %.2f, %.2f\nmax: %.2f, %.2f, %.2f\n", minV.x(), minV.y(), minV.z(), maxV.x(), maxV.y(), maxV.z());
        
}

__device__ bool Mesh::AnyHit(const Ray& ray) {
        
    vec3 nLocal = ray.invorig - ray.invdir * (minV + maxV) / 2.f;

    vec3 k = vec3(abs(ray.invdir.x()), abs(ray.invdir.y()), abs(ray.invdir.z())) * (maxV - minV) / 2.f;
    vec3 t1 = -nLocal - k;
    vec3 t2 = -nLocal + k;

    double tNear = max(max(t1.x(), t1.y()), t1.z());
    double tFar = min(min(t2.x(), t2.y()), t2.z());


    return !(tNear > tFar || tFar < 0);
        

    /*
    vec3 tMin = (minV - ray.origin) / ray.direction;
    vec3 tMax = (maxV - ray.origin) / ray.direction;
    vec3 t1 = min(tMin, tMax);
    vec3 t2 = max(tMin, tMax);
    float tNear = max(max(t1.x(), t1.y()), t1.z());
    float tFar = min(min(t2.x(), t2.y()), t2.z());

    tMaxA = tFar;

    return tNear > tFar;
    */

    //return true;
}

__device__ bool Mesh::TryHit(const Ray& ray, HitResult& closestHit) {
    bool bHit = false;
    Triangle* closestTri = nullptr;
    float finalU, finalV;

    for (int i = 0; i < size; i++) {
        Triangle* triHere = triBuffer[i];
   
        float t = 0.f;
        float u = 0.f;
        float v = 0.f;

        if (rayTriangleIntersect(ray.origin, ray.direction, triHere->v1, triHere->v2, triHere->v3, t, u, v) && t > kEpsilon && t < closestHit.t) {
            closestHit.t = t;
            closestHit.u = (1.f - u - v) * triHere->u1 + u * triHere->u2 + v * triHere->u3;
            closestHit.v = (1.f - u - v) * triHere->vt1 + u * triHere->vt2 + v * triHere->vt3;
                
            // Account for >1 and <1 UVs
            closestHit.u -= floorf(closestHit.u);
            closestHit.v -= floorf(closestHit.v);

            // Interpolate vertex normals (we are dealing with source assets.. pretty low poly)

            closestHit.HitNormal = (1.f - closestHit.u - closestHit.v) * triHere->n1 + closestHit.u * triHere->n2 + closestHit.v * triHere->n3;
            closestHit.GeometricNormal = closestHit.HitNormal;

            closestHit.objId = objectID;

            closestHit.RealU = u;
            closestHit.RealV = v;

            finalU = u;
            finalV = v;

            bHit = true;
            closestTri = triHere;
        }
    }

    if (closestTri != nullptr && bHit) {
        if (pbrMaps.normalMap.initialized) {
            // Proceed with normal mapping
            vec3 bitangent = (1.f - closestHit.u - closestHit.v) * closestTri->bin1 + closestHit.u * closestTri->bin2 + closestHit.v * closestTri->bin3;
            vec3 tangent = (1.f - closestHit.u - closestHit.v) * closestTri->tan1 + closestHit.u * closestTri->tan2 + closestHit.v * closestTri->tan2;

            glm::mat3 tbnMatrix(
                tangent.toGLM(),
                bitangent.toGLM(),
                closestHit.HitNormal.toGLM()
            );

            vec3 thisNormal = pbrMaps.normalMap.GetPixel(closestHit.u, closestHit.v) * 2.0f - 1.0;
            glm::vec3 worldSpaceNormal = tbnMatrix * thisNormal.toGLM();

            thisNormal = vec3(worldSpaceNormal.x, worldSpaceNormal.y, worldSpaceNormal.z);
            thisNormal.make_unit_vector();

            closestHit.HitNormal = thisNormal;
        }

        /*
        if (pbrMaps.mraoMap.initialized) {
            closestHit.MRAO = pbrMaps.mraoMap.GetPixel(closestHit.u, closestHit.v);
        }

        closestHit.HitAlbedo = GetColor(closestHit);

        if (pbrMaps.emissionMap.initialized) {
            vec3 emissionColorHere = pbrMaps.emissionMap.GetPixel(closestHit.u, closestHit.v);

            closestHit.HitAlbedo += emissionColorHere * emission;
        }
        */
    }

    return bHit;
}
