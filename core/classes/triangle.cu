﻿#include <classes/triangle.cuh>
#include <classes/vec3.cuh>

#include "hip/hip_runtime.h"
#include "math.h"

__device__ Triangle::Triangle(const TrianglePayload& payload) {
	v1 = payload.v1, v2 = payload.v2, v3 = payload.v3;
	u1 = payload.u1, u2 = payload.u2, u3 = payload.u3;
	vt1 = payload.vt1, vt2 = payload.vt2, vt3 = payload.vt3;

	n1 = payload.n1, n2 = payload.n2, n3 = payload.n3;
	bin1 = payload.bin1, bin2 = payload.bin2, bin3 = payload.bin3;
	tan1 = payload.tan1, tan2 = payload.tan2, tan3 = payload.tan3;
}

__device__ Triangle::Triangle() {
	v1 = vec3();
	v2 = vec3();
	v3 = vec3();

	u1 = 0.0f;
	u2 = 0.0f;
	u3 = 0.0f;

	vt1 = 0.0f;
	vt2 = 0.0f;
	vt3 = 0.0f;

	n1 = vec3();
	n2 = vec3();
	n3 = vec3();

	bin1 = vec3();
	bin2 = vec3();
	bin3 = vec3();

	tan1 = vec3();
	tan2 = vec3();
	tan3 = vec3();
}
