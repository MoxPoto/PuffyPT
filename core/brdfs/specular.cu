#include "hip/hip_runtime.h"
﻿#include <brdfs/specular.cuh>
#include <brdfs/lambert.cuh>

#include <classes/vec3.cuh>
#include <classes/ray.cuh>
#include <classes/hitresult.cuh>
#include <classes/object.cuh>

#include "hiprand/hiprand_kernel.h"
#include "hip/hip_math_constants.h"

#include <util/macros.h>

#include <math/ggx.cuh>
#include <math/basic.cuh>
#define PROTECTZERO(statement) fmaxf(0.001f, statement)

__device__ static vec3 coloredSchlick(vec3 r0, float cosine, float ref_idx) {
	return r0 + (vec3(1.0f) - r0) * powf(fmaxf((1.0f - cosine), 0), 5.0f);
}

__device__ static float FalcorNDFGGX(float alpha, float cosTheta) {
	float a2 = alpha * alpha;
	float d = ((cosTheta * a2 - cosTheta) * cosTheta + 1);
	return a2 / (d * d * static_cast<float>(HIP_PI));
}

namespace SpecularBRDF {
	__device__ vec3 reflect(const vec3& direction, const vec3& normal) {
		return direction - 2.0f * dot(direction, normal) * normal;
	}
	
	__device__ float schlick(float cosine, float ref_idx) {
		float r0 = (ref_idx - 1.0f) / (ref_idx + 1.0f);
		r0 = r0 * r0;
		return r0 + (1.0f - r0) * powf((1.0f - cosine), 5.0f);
	}

	__device__ void Eval(float alpha, float metalness, Object* target, const vec3& normal, const vec3& wo, const vec3& wi, const vec3& albedo, vec3& attenuation, float& pdf) {
		vec3 m = unit_vector((wi + wo));

		float thetaM = thetaFromVec(m);

		pdf = GGXDistribution(alpha, thetaM, normal, m) * fabsf(dot(m, normal));

		// evaluate cook-torrance

		float f = fabsf((1.f - target->lighting.ior) / (1.f + target->lighting.ior));
		// in my schlick's function, f0 is represented as r0
		float F0 = (f * f);

		vec3 finalSchlicksInput = lerpVectors(vec3(F0), albedo, metalness);

		vec3 fresnelTerm = coloredSchlick(finalSchlicksInput, dot(wo, m), target->lighting.ior);
		/*
		vec3 numerator = fresnelTerm * GGXDistribution(alpha, thetaM, normal, m) * GGXGeometry(wi, normal, m, alpha);
		attenuation = numerator;
		*/

	}

	__device__ bool SampleWorld(const HitResult& res, hiprandState* local_rand_state, float extraRand, float& pdf, const Ray& previousRay, vec3& attenuation, Ray& targetRay, Object* target) {
		// wo = -previousRay.direction;
		// wi = reflect(-wo, hitnormal);

		// some of the code is based on: https://github.com/100PXSquared/gmod-binary-tracer/blob/56f482c041909494497d22dcf5c45d4f507aa022/Binary%20Module/shaders.cpp#L63

		vec3 wo = -previousRay.direction;
		vec3 wi = reflect(previousRay.direction, res.HitNormal);
		float u1 = hiprand_uniform(local_rand_state);
		float u2 = hiprand_uniform(local_rand_state);
		// random 1 and 2 in the cook-torrance paper
		float metalness = target->lighting.metalness;
		float roughness = target->lighting.roughness;

		if (target->pbrMaps.mraoMap.initialized) {
			metalness = res.MRAO.b();
			roughness = res.MRAO.g();

			// Might be a good idea in the future to choose a specific
			// MRAO format before just assuming that metalness = b and roughness = g
		}

		float alpha = fmaxf(0.001f, roughness * roughness);
		static const float kMinCosTheta = 1e-4f;

		float thetaM = atanf((alpha * sqrtf(u1)) / sqrt(1.f - u1));
		float phiM = (2.f * static_cast<float>(HIP_PI) * u2);

		vec3 m = TransformToWorld(sinf(thetaM) * cosf(phiM), sinf(thetaM) * sinf(phiM), cosf(thetaM), res.HitNormal);
		m.make_unit_vector();

		if (dot(wo, m) < kMinCosTheta) {
			return false;
		}

		targetRay.origin = res.HitPos + (res.HitNormal * 0.01f);
		targetRay.direction = (2.f * PROTECTZERO(fabsf(dot(wo, m))) * m - wo);

		// pdf = D(m)|m * n|

		
		pdf = GGXDistribution(alpha, thetaM, res.HitNormal, m) * fabsf(dot(m, res.HitNormal));
			
		// evaluate cook-torrance

		vec3 hr = sign(dot(wi, res.HitNormal)) * (wi + wo);
		hr.make_unit_vector();

		float f = fabsf((1.f - target->lighting.ior) / (1.f + target->lighting.ior));
		// in my schlick's function, f0 is represented as r0
		float F0 = (f * f);
		vec3 finalSchlicksInput = lerpVectors(vec3(F0), res.HitAlbedo, metalness);
		vec3 fresnelTerm = coloredSchlick(finalSchlicksInput, dot(wi, m), target->lighting.ior);


		vec3 numerator = fresnelTerm * GGXDistribution(alpha, thetaM, res.HitNormal, m) * GGXGeometry(wi, wo, m, res.HitNormal, alpha);
		// float denominator = 4.f * (dot(wi, res.HitNormal)) * (dot(wo, res.HitNormal));
		
		attenuation = numerator;


		

		//Eval(alpha, metalness, target, res.HitNormal, wo, wi, res.HitAlbedo, attenuation, pdf);
		// frensel term being wacky..
		return true;
	}

	__device__ float PDF(const HitResult& res, Object* target, const vec3& wo, const vec3& wi) {
		float alpha = target->lighting.roughness;
		static const float kMinCosTheta = 1e-6f;

		if (target->pbrMaps.mraoMap.initialized) {
			alpha = res.MRAO.b();
		}

		if (min(wo.z(), wi.z()) < kMinCosTheta)
			return 0.f;

		vec3 h = unit_vector(wo + wi);
		float woDotH = dot(wo, h);

		float pdf = FalcorNDFGGX(alpha, h.z());
	    // return GGXDistribution(alpha, h.z(), res.HitNormal, h) * fabsf(dot(h, res.HitNormal));

		return pdf / (4.0 * woDotH);
	}
}


