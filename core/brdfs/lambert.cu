#include "hip/hip_runtime.h"
﻿#include <brdfs/lambert.cuh>
#include <classes/vec3.cuh>
#include <classes/ray.cuh>
#include <classes/hitresult.cuh>
#include <classes/object.cuh>

#include "hiprand/hiprand_kernel.h"
#include "math.h"
#include "hip/hip_math_constants.h"

#include <util/macros.h>

#define RANDVEC3 vec3(fmodf(hiprand_uniform(local_rand_state) + extraRand, 1.f),fmodf(hiprand_uniform(local_rand_state) + extraRand, 1.f),fmodf(hiprand_uniform(local_rand_state) + extraRand, 1.f))
//#define RANDVEC3 vec3(hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state))

static constexpr float M_1_PI = 0.318309886183790671538f;

__device__ static inline vec3 lerpVectors(vec3 a, vec3 b, float f)
{
	return (a * (1.0f - f)) + (b * f);
}

// from: https://computergraphics.stackexchange.com/questions/4979/what-is-importance-sampling/4993
__device__ static vec3 TransformToWorld(const float& x, const float& y, const float& z, const vec3& normal)
{
	// Find an axis that is not parallel to normal
	vec3 majorAxis;
	if (fabsf(normal.x()) < 0.57735026919f /* 1 / sqrt(3) */) {
		majorAxis = vec3(1, 0, 0);
	}
	else if (fabsf(normal.y()) < 0.57735026919f /* 1 / sqrt(3) */) {
		majorAxis = vec3(0, 1, 0);
	}
	else {
		majorAxis = vec3(0, 0, 1);
	}

	// Use majorAxis to create a coordinate system relative to world space
	vec3 u = unit_vector(cross(normal, majorAxis));
	vec3 v = cross(normal, u);
	vec3 w = normal;

	// Transform from local coordinates to world coordinates
	return u * x + v * y + w * z;
}

// from: https://computergraphics.stackexchange.com/questions/4979/what-is-importance-sampling/4993
__device__ static float getLambertPDF(vec3 inputDir, vec3 normal) {
	return dot(inputDir, normal) * M_1_PI;
}

// from: https://computergraphics.stackexchange.com/questions/4979/what-is-importance-sampling/4993
__device__ static vec3 evaluateLambert(vec3 inputDir, vec3 normal, vec3 albedo) {
	return albedo * M_1_PI * dot(inputDir, normal);
}

namespace LambertBRDF {
	__device__ vec3 random_in_unit_sphere(hiprandState* local_rand_state, float extraRand) {
		vec3 p;
		
		do {
			p = 2.0f * RANDVEC3 - vec3(1, 1, 1);
		} while (p.squared_length() >= 1.0f);
		return p;
	}
		
	__device__ void Eval(const vec3& normal, const vec3& wo, const vec3& wi, const vec3& albedo, vec3& attenuation, float& pdf) {
		attenuation = evaluateLambert(wi, normal, albedo);
		pdf = getLambertPDF(wi, normal);
	}

	__device__ void SampleWorld(const HitResult& rec, hiprandState* local_rand_state, float extraRand, float& pdf, vec3& attenuation, Ray& targetRay, Object* target) {
		vec3 BLACK = vec3(0.f);

		targetRay.origin = rec.HitPos + (rec.HitNormal * 0.001f);

		float r1 = hiprand_uniform(local_rand_state);
		float r2 = hiprand_uniform(local_rand_state);

		float r = sqrtf(r1);
		float theta = r2 * 2.f * HIP_PI;

		float x = r * cosf(theta);
		float y = r * sinf(theta);

		// Project z up to the unit hemisphere
		float z = sqrt(1.0f - x * x - y * y);

		vec3 sampleLocalized = TransformToWorld(x, y, z, rec.HitNormal);
		targetRay.direction = sampleLocalized;

		// Specular metallics and diffuse metallics come from: https://github.com/NVIDIAGameWorks/Falcor/blob/master/Source/Falcor/Scene/Shading.slang#L185
		float metalness = target->lighting.metalness;

		if (target->pbrMaps.mraoMap.initialized) {
			metalness = rec.MRAO.b();
		}
			
		vec3 albedo = lerpVectors(rec.HitAlbedo, BLACK, metalness);
		vec3 wo = vec3(0, 0, 0); // not used

		Eval(rec.HitNormal, wo, sampleLocalized, albedo, attenuation, pdf);
	}

	__device__ float PDF(const HitResult& res, Object* target, const vec3& wo, const vec3& wi) {
		static const float kMinCosTheta = 1e-6f;

		if (min(wo.z(), wi.z()) < kMinCosTheta)
			return 0;

		return getLambertPDF(wi, res.HitNormal);
	}
}
