#include "hip/hip_runtime.h"
﻿#include <classes/vec3.cuh>
#include <images/texture.cuh>

#include <util/macros.h>
#include <dxhook/mainHook.h>

#include <map>
#include <string>

#include "hip/hip_runtime.h"

#define checkCudaErrors(val) DXHook::check_cuda( (val), #val, __FILE__, __LINE__ )

std::map<std::string, Pixel*> deviceTextures;

__device__ Texture::Texture() {
	resX = 0;
	resY = 0;
	imageData = NULL;
	fallbackColor = vec3(0, 0, 0);
}

__device__ void Texture::SetFallbackColor(vec3 newColor) {
	fallbackColor = newColor;
}

__device__ void Texture::Initialize(int newResX, int newResY, Pixel* newImageData) {
	if (newImageData == nullptr)
		return;

	resX = newResX;
	resY = newResY;
	imageData = newImageData;
	initialized = true;
}

__device__ vec3 Texture::GetPixel(float u, float v) {
	if (imageData == nullptr)
		return fallbackColor;

			
	int x = static_cast<int>(fminf(u * resX, resX - 1));
	int y = static_cast<int>(fminf(v * resY, resY - 1));


	int base_index = (3 * (y * resY + x));

	return vec3(imageData[base_index], imageData[base_index + 1], imageData[base_index + 2]);
}

// texture management

__host__ bool IsTextureCached(const std::string& textureName) {
	auto foundValue = deviceTextures.find(textureName);

	return foundValue != deviceTextures.end();
}

__host__ Pixel* RetrieveCachedTexture(const std::string& textureName) {
	if (!IsTextureCached(textureName))
		return nullptr;

	// dev in this case stands for device
	Pixel* devTexture = nullptr;

	try {
		devTexture = deviceTextures.at(textureName);
	}
	catch (std::exception& exception) {
		HOST_DEBUG("Hit a exception; exception reads: %s", exception.what());
		
		return nullptr;
	}

	return devTexture;
}

__host__ Pixel* CreateTextureOnDevice(Pixel* hostPtr, const std::string& textureName, size_t textureSize) {
	if (IsTextureCached(textureName)) {
		return RetrieveCachedTexture(textureName);
	}

	Pixel* devPtr;

	checkCudaErrors(hipMalloc((void**)&devPtr, textureSize));
	checkCudaErrors(hipMemcpy(devPtr, hostPtr, textureSize, hipMemcpyHostToDevice));

	deviceTextures[textureName] = devPtr;

	HOST_DEBUG("Successfully created texture %s on GPU!", textureName.c_str());

	return devPtr;
}
