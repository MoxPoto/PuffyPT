#include "hip/hip_runtime.h"
﻿#include <images/hdri.cuh>
#include <images/hdriUtility.cuh>

#include <iostream>
#include <vector>
#include <filesystem>

#include <vendor/stb_image.h>
#include <util/macros.h>
#include <dxhook/mainHook.h>

#include "hip/hip_runtime.h"

#define HDRI_RESX 1024
#define HDRI_RESY 512

__global__ void createHDRIGPU(HDRI* targetHDRI, float* imageData, int resX, int resY) {

    if (imageData == nullptr) {
        NULLPTR_HIT("createHDRIGPU: hit a nullptr on imageData!!");
    }

    // (targetHDRI)->loadData(imageData);
    (targetHDRI)->resX = resX;
    (targetHDRI)->resY = resY;
    (targetHDRI)->brightness = 1.f; // something for me to remember; malloc does not invoke my constructor..
}

__host__ bool LoadHDRI(const std::string& path) {
    int width = HDRI_RESX;
    int height = HDRI_RESY;
    int comps = 3;
    float* hdriImg = stbi_loadf(path.c_str(), &width, &height, &comps, 3);

    size_t imageSize = 3 * (width * height) * sizeof(float);

    if (hdriImg != NULL) {
        HOST_DEBUG("Loaded HDRI, copying to VRAM..");
            
        checkCudaErrors(hipMemcpy(DXHook::hdriData, hdriImg, imageSize, hipMemcpyHostToDevice));
        HOST_DEBUG("Done, instantiating HDRI on gpu now..");

        createHDRIGPU << <1, 1 >> > (DXHook::mainHDRI, DXHook::hdriData, width, height);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        HOST_DEBUG("HDRI created on gpu with image data intact, continuing setup..");
    }
    else {
        NULLPTR_HIT("Hit nullptr on hdriImg!!");
        return false;
    }

    stbi_image_free(hdriImg);

    return true;
}

