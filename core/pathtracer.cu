#include "hip/hip_runtime.h"
﻿#include <pathtracer.cuh>

#include "hip/hip_runtime.h"

#include <classes/vec3.cuh>
#include <classes/ray.cuh>
#include <classes/object.cuh>

#include <dxhook/mainHook.h>

#include <brdfs/lambert.cuh>
#include <brdfs/specular.cuh>
#include <brdfs/refraction.cuh>
#include <brdfs/mixed.cuh>

#include "hip/hip_math_constants.h"

#pragma region Utility


__device__ static inline vec3 lerpVectors(vec3 a, vec3 b, float f)
{
    return (a * (1.0f - f)) + (b * f);
}

__device__ static void adjustShadingNormal(HitResult& closestHit, vec3 dir) {
    vec3 Ng = closestHit.GeometricNormal;
    vec3 Ns = closestHit.HitNormal;

    const float kCosThetaThreshold = 0.1f;
    float cosTheta = dot(dir, Ns);

    if (cosTheta <= kCosThetaThreshold) {
        float t = __saturatef(cosTheta * (1.f / kCosThetaThreshold));
        closestHit.HitNormal = lerpVectors(Ng, Ns, t);
    }
}


__device__ Object* traceScene(int count, Object** world, const Ray& ray, HitResult& output, bool aabbOverride) {
    Object* hitObject = NULL;

    output.t = FLT_MAX;

    for (int i = 0; i < count; i++) {
        Object* target = *(world + i);

        if (i == ray.ignoreID) continue;

        if (target->AnyHit(ray)) {
            // ok, then we trace the precise mesh

            if (target->TryHit(ray, output)) {
                hitObject = target;
            }
        }
    }

    // Fix our shading normal and compute HitPos
    if (hitObject != NULL) {
        output.HitPos = ray.origin + (ray.direction * output.t);

        bool inverted = dot(ray.direction, output.GeometricNormal) > 0.f;
        output.backface = inverted;

        if (inverted) {
            output.HitNormal = -output.HitNormal;
            output.GeometricNormal = -output.GeometricNormal;
        }
        
        if (hitObject->pbrMaps.mraoMap.initialized) {
            output.MRAO = hitObject->pbrMaps.mraoMap.GetPixel(output.u, output.v);
        }

        output.HitAlbedo = hitObject->GetColor(output);

        if (hitObject->pbrMaps.emissionMap.initialized) {
            vec3 emissionColorHere = hitObject->pbrMaps.emissionMap.GetPixel(output.u, output.v);

            output.HitAlbedo += emissionColorHere * hitObject->emission;
        }
        
        // adjustShadingNormal(output, ray.direction);
    }

    return hitObject;
}

__device__ vec3 genSkyColor(HDRI* mainHDRI, SkyInfo skyInfo, float* imgData, const vec3& dir) {
    /*
    float t = 0.5f * (dir.z() + 1.0f);
    vec3 skyColor = (1.0f - t) * skyInfo.azimuth + t * skyInfo.zenith;
    */

    vec3 skyColor = mainHDRI->GetPixelFromRay(dir, imgData);

    return skyColor;
}

#pragma endregion Utility

#pragma region Shading

static __device__ const int EMISSIVE_MINIMUM = 15; // Minimum emission to be considered a light
#define min(a,b) ((a)<(b)?(a):(b))
#define max(a,b) ((a)>(b)?(a):(b))

__device__ vec3 calcDirect(int count, Object** world, Object* firstHit, const Ray& ray, const HitResult& rec) {


    vec3 lightObtained(0, 0, 0);
    int lightHits = 0;

    for (int i = 0; i < count; i++) {
        Object* light = *(world + i);

        if (light->emission >= EMISSIVE_MINIMUM) {
            float lightPower = 300.f + ((light->emission - EMISSIVE_MINIMUM) * 2.f); // The more intense emission is, more range is added
            float lightBrightness = 1.f;

            vec3 newOrigin = rec.HitPos + (rec.HitNormal * 0.001f);
            vec3 testDirection = unit_vector((light->position - newOrigin));

            Ray testRay(newOrigin, testDirection);
            HitResult testResult;

            Object* hitObject = traceScene(count, world, testRay, testResult);

            // A path from the sampled position and the light has been found
            if (hitObject != NULL && hitObject->objectID == light->objectID && testResult.t <= lightPower) {
                // float normalizedRange = (distance / lightPower);

                float falloff = lightPower / ((0.01 * 0.01) + powf(testResult.t, 2.f));

                vec3 lightContribution = (light->GetColor(testResult) * falloff) * lightBrightness;

                lightHits++;
                lightObtained += lightContribution;
            }
        }
    }

    if (lightHits == 0) {
        return lightObtained;
    }
    else {
        lightObtained /= static_cast<float>(lightHits);
        return lightObtained;
    }

}

static __device__ PathtraceResult depthColor(DXHook::RenderOptions* options, const Ray& ray, hiprandState* local_rand_state) {
    Ray cur_ray = ray;
    vec3 currentLight(1, 1, 1);
    
    PathtraceResult res;
    res.vertices = options->max_depth + 1;
    res.eyePath = reinterpret_cast<LightHit*>(malloc(sizeof(LightHit) * res.vertices));

    res.vertices = 0;

    for (int i = 0; i < options->max_depth; i++) {
        HitResult rec;
        Object* target = traceScene(options->count, options->world, cur_ray, rec);

        if (target != NULL) {
            // set our current ray to the new formulated one (this being perfect diffuse)
            // and attenuate our color by the albedo we hit, but we also should multiply our albedo by the objects emission

            if (!target->pbrMaps.emissionMap.initialized && target->emission > EMISSIVE_MINIMUM) {
                // just return the light
                LightHit hitPoint;
                hitPoint.hitPos = rec.HitPos;
                hitPoint.startPos = cur_ray.origin;

                hitPoint.attenuation = (target->GetColor(rec) * target->emission);
                hitPoint.pdf = 1.f;

                hitPoint.isLight = true;

                res.eyePath[res.vertices++] = hitPoint;
                res.color = currentLight * (target->GetColor(rec) * target->emission);
                return res;
            }

            Ray new_ray(vec3(0, 0, 0), vec3(0, 0, 0));
            vec3 attenuation = currentLight;
            float pdf = 1.f;

            /*
            switch (target->matType) {
            case (BRDF::Lambertian):
                LambertBRDF::SampleWorld(rec, local_rand_state, options->curtime, pdf, attenuation, new_ray, target);
                break;
            case (BRDF::Specular):
                SpecularBRDF::SampleWorld(rec, local_rand_state, options->curtime, pdf, cur_ray, attenuation, new_ray, target);
                break;
            case (BRDF::Refraction):
                RefractBRDF::SampleWorld(rec, local_rand_state, pdf, options->curtime, cur_ray, attenuation, new_ray, target);
                break;
            default:
                break;
            }
            */

            LightHit thisHit;
            thisHit.hitPos = rec.HitPos;
            thisHit.startPos = cur_ray.origin;
            thisHit.isLight = false;

            bool validSample = MixedBxDF::SampleWorld(rec, local_rand_state, options->curtime, pdf, attenuation, cur_ray, new_ray, target, thisHit.brdf);

            if (!validSample) {
                // Nothing was chosen from our BxDF, so continue onwards
                continue;
            }

            thisHit.attenuation = attenuation;
            thisHit.pdf = pdf;

            currentLight *= attenuation / pdf;

            // russian roulette to terminate paths that barely contain any visible contribution
            // from: https://computergraphics.stackexchange.com/a/5808

            /*
            float prob = max(currentLight.x(), max(currentLight.y(), currentLight.z()));

            if (hiprand_uniform(local_rand_state) > prob) {
                return currentLight;
            }

            // ok, now we add the energy lost from russian rouletting:
            currentLight *= 1 / prob;
            */

            cur_ray = new_ray;

            res.eyePath[res.vertices++] = thisHit;
            
        }
        else {
            // didnt hit, finish our depth trace by attenuating our final hit color by the sky color

            if (options->doSky) {
                LightHit thisHit;
                thisHit.hitPos = rec.HitPos;
                thisHit.isLight = true;

                vec3 skyColor = genSkyColor(options->hdri, options->skyInfo, options->hdriData, cur_ray.direction);

                thisHit.attenuation = skyColor;
                thisHit.pdf = 1.f;
                
                res.color = (currentLight * (skyColor));
                res.eyePath[res.vertices++] = thisHit;

                return res;
            }
            else {
                LightHit thisHit;
                thisHit.hitPos = rec.HitPos;
                thisHit.isLight = true;
                
                const vec3 thisSkyColor = vec3(0.3, 0.3, 0.3);
                
                thisHit.attenuation = thisSkyColor;
                thisHit.pdf = 1.f;

                res.color = (currentLight * thisSkyColor);
                res.eyePath[res.vertices++] = thisHit;

                return res;
            }
        }
    }

    res.color = vec3(0.f, 0.f, 0.f);
    return res; // exceeded recursion..
}

__device__ PathtraceResult pathtrace(DXHook::RenderOptions* options, const Ray& ray, hiprandState* local_rand_state) {
    vec3 indirectLighting(0, 0, 0);
    vec3 directLighting(0, 0, 0);

    PathtraceResult res;

    HitResult result;
    Object* hitObject = traceScene(options->count, options->world, ray, result);

    if (hitObject != NULL) {
        directLighting = calcDirect(options->count, options->world, hitObject, ray, result);
    }

    for (int i = 0; i < options->samples; i++) {
        PathtraceResult depthRes = depthColor(options, ray, local_rand_state);
        indirectLighting += depthRes.color;
        res.eyePath = depthRes.eyePath;
        res.vertices = depthRes.vertices;
    }

    indirectLighting /= (float)options->samples;

    if (options->curPass == 0) { // Direct only
        res.color = directLighting;
    }
    else if (options->curPass == 1) { // Indirect only
        res.color = indirectLighting;
    }
    else {
        res.color = (directLighting / HIP_PI + 2.0 * indirectLighting);
    }

    return res;
}

#pragma endregion Shading