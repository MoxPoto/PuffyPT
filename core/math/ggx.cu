#include "hip/hip_runtime.h"
﻿// GGX Definitions
#include "hip/hip_runtime.h"
#include <classes/vec3.cuh>
#include <math/ggx.cuh>

#include "hip/hip_math_constants.h"

__device__ float thetaFromVec(vec3 vec) {
	return atanf(sqrtf(vec.x() * vec.x() + vec.y() * vec.y()) / vec.z());
}

// χ+(a)
__device__ float chi(float num) {
	return num > 0.f ? 1.f : 0.f;
}

// D(m)
__device__ float GGXDistribution(float width, float thetaM, const vec3& hitNormal, const vec3& microfacet) {
	float alphaSquared = (width * width);

	float numerator = alphaSquared * chi(dot(microfacet, hitNormal));
	float tanThetaM = tanf(thetaM);
	float term2 = (alphaSquared + powf(tanThetaM, 2));

	float denominator = (static_cast<float>(HIP_PI) * powf(cosf(thetaM), 4.f) * (term2 * term2));

	return numerator / denominator;
}

// G1(v, m)
__device__ float GGXGeometry(const vec3& v, const vec3& n, const vec3& m, float width) {
	float vdotm = dot(v, m);
	float vdotn = dot(v, n);

	float chiOfDot = chi(vdotm / vdotn);
	float alphaSquared = (width * width);
	float thetaOfV = thetaFromVec(v);
	float tanPart = tanf(thetaOfV);
	float denominator = 1.f + sqrtf(1.f + (alphaSquared * (tanPart * tanPart)));

	return chiOfDot * (2.f / denominator);
}