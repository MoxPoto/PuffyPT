#include "hip/hip_runtime.h"
﻿#include "specular.cuh"
#include "lambert.cuh"

#include "../vec3.cuh"
#include "../ray.cuh"
#include "../hitresult.cuh"
#include "../object.cuh"

#include "hiprand/hiprand_kernel.h"

namespace Tracer {
	namespace SpecularBRDF {
		__device__ vec3 reflect(const vec3& direction, const vec3& normal) {
			return direction - 2.0f * dot(direction, normal) * normal;
		}

		__device__ float schlick(float cosine, float ref_idx) {
			float r0 = (1.0f - ref_idx) / (1.0f + ref_idx);
			r0 = r0 * r0;
			return r0 + (1.0f - r0) * powf((1.0f - cosine), 5.0f);
		}

		__device__ void SampleWorld(const HitResult& res, hiprandState* local_rand_state, const Ray& previousRay, vec3& attenuation, Ray& targetRay, Object* target) {
			targetRay.origin = res.HitPos;
			targetRay.direction = reflect(previousRay.direction, res.HitNormal);

			attenuation = (target->color * target->emission);

			if (target->lighting.roughness > 0.05f) {
				vec3 sampleDir = target->lighting.roughness * LambertBRDF::random_in_unit_sphere(local_rand_state);

				targetRay.direction = unit_vector(targetRay.direction + sampleDir);
			}
		}
	}
}

