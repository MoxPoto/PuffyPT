﻿#include <pathtracer/pathtracer.cuh>
#include <renderer/render.cuh>

#include <hip/hip_runtime.h>
#include <imgui.h>

#include <iostream>

__host__ void Pathtracer::Update() {
    updateMutex.lock();
    int tileX = 6;
    int tileY = 6;

    dim3 blocks(width / tileX + 1, height / tileY + 1);
    dim3 threads(tileX, tileY);

    renderKernel << <blocks, threads >> > (framebuffer, dxFramebuffer, width, height);
    
    updateMutex.unlock();
}

__host__ void Pathtracer::ImGuiUpdate() {
    ImGui::Text("FPS: %.1f", ImGui::GetIO().Framerate);
}

__host__ void Pathtracer::ErrorCheck(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cout << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n" << "CUDA_ERROR_STRING: " << hipGetErrorString(result) << "\n" <<
            hipGetErrorName(result) << "\n";
    }
}

__host__ void Pathtracer::Allocate(void* gpuMemory, size_t bufferSize, bool managed) {
    if (managed) {
        checkCudaErrors(hipMallocManaged((void**)&gpuMemory, bufferSize));
    }
    else {
        checkCudaErrors(hipMalloc((void**)&gpuMemory, bufferSize));
    }

    buffersToRelease.push_back(gpuMemory);
}

__host__ Pathtracer::Pathtracer(int _width, int _height) {
    width = _width;
    height = _height;

    size_t num_pixels = static_cast<size_t>(width) * static_cast<size_t>(height);
    Allocate(framebuffer, num_pixels * 3 * sizeof(float), true);
    Allocate(dxFramebuffer, num_pixels * sizeof(DWORD), true);

    valid = true;
}

__host__ Pathtracer::~Pathtracer() {
    updateMutex.lock();
    // Release all the buffers we allocated
    for (void* buffer : buffersToRelease) {
        hipFree(buffer);
    }
    
    valid = false; // Invalidate the class
    updateMutex.unlock();
}