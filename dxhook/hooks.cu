#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"
#include <d3dx9math.h>

#include "mainHook.h"
#include <d3d9.h>
#include <imgui_impl_dx9.h>
#include <imgui_impl_win32.h>
#include <Windows.h>
#include <iostream>
#include <string>
#include <chrono>

#include "../ray.cuh"
#include "../mesh.cuh"
#include "../vec3.cuh"
#include "../object.cuh"
#include "../triangle.cuh"
#include "../postprocess/mainDenoiser.cuh"
#include <chrono>
#include <random>

#include "../images/hdriUtility.cuh"


#define WIDTH 480
#define HEIGHT 270
#define checkCudaErrors(val) DXHook::check_cuda( (val), #val, __FILE__, __LINE__ )
#define HDRI_FOLDER "C:\\pathtracer\\hdrs"
#define PUFF_INCREMENT(name, variable) ImGui::Button(name); if (ImGui::IsItemActive()) { variable += 0.1f; }
#define PUFF_DECREMENT(name, variable) ImGui::Button(name); if (ImGui::IsItemActive()) { variable -= 0.1f; }

#define PUFF_INCREMENT_RESET(name, variable) ImGui::Button(name); if (ImGui::IsItemActive()) { variable += 0.1f; frameCount = 0;}
#define PUFF_DECREMENT_RESET(name, variable) ImGui::Button(name); if (ImGui::IsItemActive()) { variable -= 0.1f; frameCount = 0;}

#define VERSION "PUFFY PT - 0.08"

struct Vertex
{
	float _x, _y, _z;
	float _nx, _ny, _nz;
	float _u, _v; // texture coordinates
	static const DWORD FVF;
};

const DWORD Vertex::FVF = D3DFVF_XYZ | D3DFVF_NORMAL | D3DFVF_TEX1;

std::default_random_engine randEngine;
std::uniform_real_distribution<float> unif(0.0, 1.0);

namespace DXHook {
	EndScene oldFunc;
	void* d3d9Device[119];
	LPDIRECT3DDEVICE9 device;
	bool gotDevice = false;

	float* fb;
	float* postFB;
	Tracer::Camera mainCam;
	Tracer::SkyInfo skyInfo;

	Tracer::Object** world;
	hiprandState* d_rand_state;
	IDirect3DTexture9* pathtraceOutput = NULL;
	IDirect3DVertexBuffer9* quadVertexBuffer = NULL;
	ID3DXSprite* pathtraceObject = NULL;
	ID3DXFont* msgFont = NULL;
	float fov = 114.f;
	int currentPass = 2;
	Tracer::vec3 camDir;

	float azimuth[3] = { 1, 1, 1 };
	float zenith[3] = { 0.5f, 0.7f, 1.0f };

	Tracer::vec3* origin;
	float curX = 0, curY = 0, curZ = 0;
	float curPitch = 0, curYaw = 0, curRoll = 0;
	Tracer::Post::GBuffer* gbufferData;
	bool denoiserEnabled = true;
	bool showSky = true;
	int world_count = 0;
	int frameCount = 0;
	bool aabbOverride = false;

	Tracer::HDRI* mainHDRI = NULL;
	float* hdriData = NULL;

	int samples = 1;
	int max_depth = 6; // less than 4 results in really, really bad reflections
	bool showPathtracer = true;
	std::chrono::steady_clock::time_point lastTime;
	float curTime = 0.f;

	int curHDRI = 0;
	std::vector<std::string> hdriList;
	int hdriListSize = 0;
	float hdriBrightness = 1.f;

	HRESULT __stdcall EndSceneHook(LPDIRECT3DDEVICE9 pDevice) {
		using Tracer::vec3;

		if (!gotDevice) {
			gotDevice = true;
			device = pDevice;

			
			HRESULT failCode = D3DXCreateTexture(device, WIDTH, HEIGHT, 1, D3DUSAGE_DYNAMIC, D3DFMT_X8R8G8B8, D3DPOOL_DEFAULT, &pathtraceOutput);

			if (!pathtraceOutput) {
				
				std::cout << "Failed to create FB texture for the pathtracer.. Code: " << failCode << "\nD3DERR_INVALIDCALL: " << 
					D3DERR_INVALIDCALL << std::endl;
			}

			HRESULT failCode2 = D3DXCreateSprite(device, &pathtraceObject);

			if (!pathtraceObject) {

				std::cout << "Failed to create sprite for the pathtracer.. Code: " << failCode2 << "\nD3DERR_INVALIDCALL: " <<
					D3DERR_INVALIDCALL << std::endl;
			}


			HRESULT failCode3 = D3DXCreateFont(
				pDevice,
				18,
				0,
				FW_NORMAL,
				1,
				FALSE,
				DEFAULT_CHARSET,
				OUT_DEFAULT_PRECIS,
				ANTIALIASED_QUALITY,
				DEFAULT_PITCH | FF_DONTCARE,
				"Terminal",
				&msgFont
			);

			if (!msgFont) {

				std::cout << "Failed to create font for the pathtracer.. Code: " << failCode3 << "\nD3DERR_INVALIDCALL: " <<
					D3DERR_INVALIDCALL << std::endl;
			}

			pDevice->CreateVertexBuffer(6 * sizeof(Vertex), D3DUSAGE_WRITEONLY, Vertex::FVF, D3DPOOL_DEFAULT, &quadVertexBuffer, 0);

			Vertex* v;
			quadVertexBuffer->Lock(0, 0, (void**)&v, 0);

			// quad built from two triangles, note texture coordinates:
			v[0] = { -1.0f, -1.0f, 1.25f, 0.0f, 0.0f, -1.0f, 0.0f, 1.0f }; // was Vertex()
			v[1] = { -1.0f, 1.0f, 1.25f, 0.0f, 0.0f, -1.0f, 0.0f, 0.0f };
			v[2] = { 1.0f, 1.0f, 1.25f, 0.0f, 0.0f, -1.0f, 1.0f, 0.0f };
			v[3] = { -1.0f, -1.0f, 1.25f, 0.0f, 0.0f, -1.0f, 0.0f, 1.0f };
			v[4] = { 1.0f, 1.0f, 1.25f, 0.0f, 0.0f, -1.0f, 1.0f, 0.0f };
			v[5] = { 1.0f, -1.0f, 1.25f, 0.0f, 0.0f, -1.0f, 1.0f, 1.0f };
			quadVertexBuffer->Unlock();

			ImGui_ImplDX9_Init(device);
		}


		HRESULT result = oldFunc(pDevice);

		D3DRASTER_STATUS rasterStatus;
		HRESULT statRes = pDevice->GetRasterStatus(0, &rasterStatus);

		ImGui_ImplWin32_NewFrame();
		ImGui_ImplDX9_NewFrame();

		ImGui::NewFrame();
		DXHook::UpdateImGUI();

		bool showWind = true;

		ImGui::SetNextWindowFocus();

	
		// test panel
		ImGui::Begin("Shader Modifier");

		ImGui::PushFont(ourFont);

		ImGui::Button("Increase FOV");

		if (ImGui::IsItemActive()) {
			fov += 1.f;
		};

		ImGui::Button("Decrease FOV");

		if (ImGui::IsItemActive()) {
			fov -= 1.f;
		}
		
		ImGui::Text("Current FOV: %.2f", fov);

		ImGui::Text("Current Samples: %d", samples);
		ImGui::Text("Current Max Depth: %d", max_depth);
		
		if (ImGui::Button("Increase Samples")) {
			samples += 5;
		}
		
		if (ImGui::Button("Decrease Samples")) {
			samples -= 5;
		}

		if (ImGui::Button("Increase Depth")) {
			max_depth += 1;
		}

		if (ImGui::Button("Decrease Depth")) {
			max_depth -= 1;
		}

		PUFF_INCREMENT("Exposure Increase", mainCam.exposure);
		PUFF_DECREMENT("Exposure Decrease", mainCam.exposure);
		ImGui::TextColored(ImVec4(1, 0, 0, 1), "HDRI Index: %d", curHDRI);
		ImGui::Text("Current HDRI: ");
		ImGui::SameLine();
		ImGui::Text(hdriList.at(curHDRI).c_str());
		
		if (ImGui::Button("Left")) {
			curHDRI = max(min(curHDRI - 1, hdriListSize - 1), 0);
		
			Tracer::LoadHDRI(hdriList.at(curHDRI));
			frameCount = 0;
		}

		if (ImGui::Button("Right")) {
			curHDRI = max(min(curHDRI + 1, hdriListSize - 1), 0);

			Tracer::LoadHDRI(hdriList.at(curHDRI));
			frameCount = 0;
		}

		if (ImGui::Button("Refresh HDRI List")) {
			Tracer::FindHDRIs(HDRI_FOLDER, hdriList, hdriListSize);
		}

		PUFF_INCREMENT_RESET("Increase HDRI Brightness", hdriBrightness);
		PUFF_DECREMENT_RESET("Decrease HDRI Brightness", hdriBrightness);

		ImGui::Checkbox("Enable Postprocessing?", &denoiserEnabled);
		ImGui::Checkbox("Show Output?", &showPathtracer);
		ImGui::Checkbox("Show Sky?", &showSky);
		ImGui::Checkbox("Override AABB Accel?", &aabbOverride);

		if (ImGui::ColorPicker3("Edit Sky Azimuth", azimuth)) {
			frameCount = 0;
		}

		if (ImGui::ColorPicker3("Edit Sky Zenith", zenith)) {
			frameCount = 0;
		}

		skyInfo.azimuth = vec3(azimuth[0], azimuth[1], azimuth[2]);
		skyInfo.zenith = vec3(zenith[0], zenith[1], zenith[2]);

		const char* passes[] = {
			"Direct Lighting",
			"Indirect Lighting",
			"Combined"
		};

		if (ImGui::ListBox("Passes", &currentPass, passes, 3))
			frameCount = 0;

		ImGui::End();

		ImGui::PopFont();

		ImGui::EndFrame();

		curTime = unif(randEngine); // hel p


		int warpX = 16;
		int warpY = 16; // technically can be ruled out as tiled rendering

		dim3 blocks(WIDTH / warpX + 1, HEIGHT / warpY + 1);
		dim3 threads(warpX, warpY);


		// std::chrono::steady_clock::time_point startTime = std::chrono::high_resolution_clock::now();

		if (showPathtracer) {
			RenderOptions options;
			options.count = world_count;
			options.fov = fov;
			options.x = curX;
			options.y = curY;
			options.z = curZ;
			options.pitch = curPitch;
			options.yaw = curYaw;
			options.roll = curRoll;
			options.frameBuffer = fb;
			options.world = world;
			options.max_x = WIDTH;
			options.max_y = HEIGHT;
			options.rand_state = d_rand_state;
			options.samples = samples;
			options.max_depth = max_depth;
			options.gbufferPtr = gbufferData;
			options.frameCount = frameCount;
			options.curtime = curTime;
			options.doSky = showSky;
			options.hdri = mainHDRI;
			options.hdriData = hdriData;
			options.curPass = currentPass;
			options.skyInfo = skyInfo;
			options.cameraDir = camDir;
			options.aabbOverride = aabbOverride;
			options.hdriBrightness = hdriBrightness;
			render << <blocks, threads >> > (options);
			checkCudaErrors(hipGetLastError());
			checkCudaErrors(hipDeviceSynchronize());

			frameCount++;

			if (denoiserEnabled) {
				Tracer::ApplyPostprocess(WIDTH, HEIGHT, blocks, threads);
			}

		}
		// std::chrono::steady_clock::time_point endTime = std::chrono::high_resolution_clock::now();
		// double timeSpent = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();

		// std::cout << "Finished rendering in " << timeSpent << " milliseconds, saving to tex\n";

		if (pathtraceOutput) {
			D3DLOCKED_RECT memRegion;
			pathtraceOutput->LockRect(0, &memRegion, NULL, D3DLOCK_DISCARD);
			// std::cout << "Updating texture pt 1\n";
			int num_pixels = WIDTH * HEIGHT;
			
			unsigned char* dest = static_cast<unsigned char*>(memRegion.pBits);
			//unsigned char* newFb = static_cast<unsigned char*>(malloc(4 * num_pixels * sizeof(unsigned char)));
			// SO GET THIS I WAS FUCKING MALLOC'ING THIS EVERY FRAME AND FORGETTING TO REMOVE IT
			// AND MY SYSTEM BSODED AND MY GPU GOT STUCK IN A RANDOM STATE LMAOOOOOO

			char* data = reinterpret_cast<char*>(memRegion.pBits);

			for (int y = 0; y < HEIGHT; ++y) {
				DWORD* row = (DWORD*)data;
				for (int x = 0; x < WIDTH; ++x) {
					int pixel_index = y * WIDTH * 3 + x * 3;
					int r = int(postFB[pixel_index] * 255.99);
					int g = int(postFB[pixel_index + 1] * 255.99);
					int b = int(postFB[pixel_index + 2] * 255.99);

					if (!denoiserEnabled) {
						r = int(fb[pixel_index] * 255.99);
						g = int(fb[pixel_index + 1] * 255.99);
						b = int(fb[pixel_index + 2] * 255.99);
					}

					*row++ = D3DCOLOR_XRGB(r, g, b);
				}
				data += memRegion.Pitch;
			}

			pathtraceOutput->UnlockRect(0);

			if (pathtraceObject) {
				D3DXMATRIX transformation;

				D3DXMatrixIdentity(&transformation);
				D3DXMatrixScaling(&transformation, 4, 4, 1);

				if (showPathtracer) {
					pathtraceObject->Begin(D3DXSPRITE_SORT_DEPTH_FRONTTOBACK);
					pathtraceObject->SetTransform(&transformation);
					pathtraceObject->Draw(pathtraceOutput, NULL, NULL, &D3DXVECTOR3(0.3, 0.3, 1), D3DCOLOR_RGBA(255, 255, 255, 255));
					pathtraceObject->End();
				}

				if (msgFont) {
					RECT msgRect;
					SetRect(&msgRect, 0, 15, 1920, 120);

					msgFont->DrawText(NULL, VERSION, -1, &msgRect, DT_CENTER | DT_NOCLIP, D3DCOLOR_ARGB(255, 10, 10, 10));


				}

				pDevice->SetRenderState(D3DRS_ZENABLE, FALSE);
				pDevice->SetRenderState(D3DRS_ALPHABLENDENABLE, FALSE);
				pDevice->SetRenderState(D3DRS_SCISSORTESTENABLE, FALSE);

				ImGui::Render();
				ImGui_ImplDX9_RenderDrawData(ImGui::GetDrawData());
			}

		}

		return result;
	}


}