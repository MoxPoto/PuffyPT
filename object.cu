﻿#include "object.cuh"
#include "ray.cuh"
#include "hitresult.cuh"
#include "vec3.cuh"

namespace Tracer {
	__host__ __device__ Object::Object() {
		color = vec3(0, 0, 0);
		emission = 1.f;
	}

	__device__ vec3 Object::getColor(const HitResult& rayThatHit) {
		/*
		if (!texture.initialized)
			return color;

		return texture.GetPixel(rayThatHit.u, rayThatHit.v) * color;
		*/

		return vec3(rayThatHit.u, rayThatHit.v, 1.f - rayThatHit.u - rayThatHit.v);
	}

	__host__ __device__ bool Object::tryHit(const Ray& ray, HitResult& result) {
		return false;
	}

	__host__ __device__ bool Object::anyHit(const Ray& ray) {
		return true; // if something simply just returns "true" on the anyhit pass it's pretty much safe to assume it's not accelerated
	}
}